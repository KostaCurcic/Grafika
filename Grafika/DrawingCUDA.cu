#include "hip/hip_runtime.h"
#include "Drawing.h"
#include "Ray.h"

#ifdef CUDA

#include <math.h>
#include <Windows.h>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#define SPHC 2
#define TRIS 3
#define LIGHTS 1

#define THRCOUNT 8

//Point camera = Point(0, 0, -2.0f);
Sphere spheres[SPHC];
Point lights[LIGHTS];
Triangle triangles[TRIS];
float angle = 0;
char *imgptr, *devImgPtr;
Sphere *devSpheres;
Point *devLights;
Triangle *devTriangles;

void InitFrame()
{
	spheres[0] = Sphere(Point(sinf(angle) * 3, -1, 10 + cosf(angle) * 3), 1);
	spheres[0].mirror = true;

	spheres[1] = Sphere(Point(5, -1, 5), 1);
	spheres[1].color.r = 50;
	spheres[1].color.g = 200;
	spheres[1].color.b = 100;

	lights[0] = Point(2, 2, 10);
	triangles[0] = Triangle(Point(10, -2, 0), Point(-10, -2, 0), Point(10, -2, 20));
	triangles[1] = Triangle(Point(-10, -2, 0), Point(-10, -2, 20), Point(10, -2, 20));

	triangles[2] = Triangle(Point(-6, 2, 6), Point(-5, -2, 8), Point(-5, -5, 4));
	triangles[2].color.r = 240;

	angle += 0.01;
}

__device__ float pointLit(Point &p, Vector n, GraphicsObject* self, Point *lights, Sphere *spheres, Triangle *triangles) {
	Ray ray;
	float lit = 0, t;
	bool col;
	for (int i = 0; i < LIGHTS; i++) {
		ray = Ray(p, lights[i]);
		if (n * ray.d > 0) {
			col = false;
			for (int j = 0; j < SPHC; j++) {
				if (spheres + j != self && ray.intersects(spheres[j], &t) && t > 0.0001) {
					col = true;
					break;
				}
			}
			if (!col) {
				for (int j = 0; j < TRIS; j++) {
					if (triangles + j != self && ray.intersects(triangles[j], &t) && t > 0.0001) {
						col = true;
						break;
					}
				}
			}
			if (!col) {
				lit += n * ray.d;
			}
		}
	}
	return lit;
}

__device__ bool findColPoint(Ray ray, Point *colPoint, Vector *colNormal, GraphicsObject **colObj, Sphere *spheres, Triangle *triangles) {

	float t1, nearest = INFINITY;
	bool mirror = false;

	for (int i = 0; i < SPHC; i++) {
		if (ray.intersects(spheres[i], &t1, nullptr)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				*colPoint = ray.getPointFromT(t1);
				*colNormal = spheres[i].Normal(*colPoint);
				*colObj = spheres + i;
				mirror = spheres[i].mirror;
			}
		}
	}

	for (int i = 0; i < TRIS; i++) {
		if (ray.intersects(triangles[i], &t1)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				*colPoint = ray.getPointFromT(t1);
				*colNormal = triangles[i].n;
				*colObj = triangles + i;
				mirror = triangles[i].mirror;
			}
		}
	}

	if (mirror) {
		return findColPoint(Ray(*colPoint, ray.d.Reflect(*colNormal)), colPoint, colNormal, colObj, spheres, triangles);
	}

	if (nearest < INFINITY) return true;
	return false;
}


__global__ void drawPixelCUDA(char* ptr, Point *lights, Sphere *spheres, Triangle *triangles) {
	int xi = blockIdx.x * THRCOUNT + threadIdx.x;
	int yi = blockIdx.y * THRCOUNT + threadIdx.y;

	if (xi > XRES || yi > YRES) return;

	float x = xi * 2.0f / YRES - XRES / (float)YRES;
	float y = yi * 2.0 / YRES - 1.0;

	char *pix = ptr + (yi * XRES + xi) * 3;

	Point pixelPoint(x, y, 0);

	Point camera = Point(0, 0, -2.0f);
	Vector normal;
	GraphicsObject *obj;

	Ray ray = Ray(camera, pixelPoint);

	float light;

	Point colPoint;

	if (findColPoint(ray, &colPoint, &normal, &obj, spheres, triangles)) {
		light = pointLit(colPoint, normal, obj, lights, spheres, triangles);
		pix[0] = obj->color.r * light;
		pix[1] = obj->color.g * light;
		pix[2] = obj->color.b * light;
	}
	else{
		pix[0] = 40;
		pix[1] = 120;
		pix[2] = 240;

	}

}

void InitDrawing(char * ptr)
{
	imgptr = ptr;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&devImgPtr, XRES * YRES * 3 * sizeof(char));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devSpheres, SPHC * sizeof(Sphere));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devLights, LIGHTS * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devTriangles, TRIS * sizeof(Triangle));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}
}

void DrawFrame()
{
	InitFrame();
	
	hipError_t cudaStatus = hipMemcpy(devSpheres, spheres, SPHC * sizeof(Sphere), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	cudaStatus = hipMemcpy(devLights, lights, LIGHTS * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	cudaStatus = hipMemcpy(devTriangles, triangles, TRIS * sizeof(Triangle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	dim3 thrds(THRCOUNT, THRCOUNT);
	dim3 blocks(XRES / THRCOUNT, YRES / THRCOUNT);

	drawPixelCUDA << <blocks, thrds >> > (devImgPtr, devLights, devSpheres, devTriangles);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(imgptr, devImgPtr, XRES * YRES * 3 * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return;
	}
}
#endif
