#include "hip/hip_runtime.h"
#include "Drawing.h"
#include "Ray.h"

#ifdef CUDA

#include <math.h>
#include <Windows.h>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#define SPHC 1
#define TRIS 2
#define LIGHTS 1

#define THRCOUNT 8

//Point camera = Point(0, 0, -2.0f);
Sphere spheres[SPHC];
Point lights[LIGHTS];
Triangle triangles[TRIS];
float angle = 0;
char *imgptr, *devImgPtr;
Sphere *devSpheres;
Point *devLights;
Triangle *devTriangles;

void InitFrame()
{
	spheres[0] = Sphere(Point(sinf(angle) * 3, -2, 10 + cosf(angle) * 3), 1);
	angle += 0.01;
	//spheres[1] = Sphere(Point(0, -1000, 10), 995);
	lights[0] = Point(2, 2, 10);
	triangles[0] = Triangle(Point(10, -2, 0), Point(-10, -2, 0), Point(10, -2, 20));
	triangles[1] = Triangle(Point(-10, -2, 0), Point(-10, -2, 20), Point(10, -2, 20));
	//lights[1] = Point(1000, 0, 0);
}

__device__ float pointLit(Point &p, Vector n, void* self, Point *lights, Sphere *spheres, Triangle *triangles) {
	Ray ray;
	float lit = 0, t;
	bool col;
	for (int i = 0; i < LIGHTS; i++) {
		ray = Ray(p, lights[i]);
		if (n * ray.d > 0) {
			col = false;
			for (int j = 0; j < SPHC; j++) {
				if (spheres + j != self && ray.intersects(spheres[j], &t) && t > 0.001) {
					col = true;
					break;
				}
			}
			if (!col) {
				for (int j = 0; j < TRIS; j++) {
					if (triangles + j != self && ray.intersects(triangles[j], &t) && t > 0.001) {
						col = true;
						break;
					}
				}
			}
			if (!col) {
				lit += n * ray.d;
			}
		}
	}
	return lit;
}


__global__ void drawPixelCUDA(char* ptr, Point *lights, Sphere *spheres, Triangle *triangles) {
	int xi = blockIdx.x * THRCOUNT + threadIdx.x;
	int yi = blockIdx.y * THRCOUNT + threadIdx.y;

	if (xi > XRES || yi > YRES) return;

	float x = xi * 2.0f / YRES - XRES / (float)YRES;
	float y = yi * 2.0 / YRES - 1.0;

	char *pix = ptr + (yi * XRES + xi) * 3;

	Point pixelPoint(x, y, 0);

	Point camera = Point(0, 0, -2.0f);
	Vector normal;
	void *obj;

	Ray ray = Ray(camera, pixelPoint);

	float t1, t2, light, nearest = INFINITY;
	Point colPoint;

	for (int i = 0; i < SPHC; i++) {
		if (ray.intersects(spheres[i], &t1, nullptr)) {
			if (t1 < nearest) {
				nearest = t1;
				colPoint = ray.getPointFromT(t1);
				normal = spheres[i].Normal(colPoint);
				obj = spheres + i;
			}
		}
	}

	for (int i = 0; i < TRIS; i++) {
		if (ray.intersects(triangles[i], &t1)) {
			if (t1 < nearest) {
				nearest = t1;
				colPoint = ray.getPointFromT(t1);
				normal = triangles[i].n;
				obj = triangles + i;
			}
		}
	}

	if (nearest < INFINITY) {
		light = pointLit(colPoint, normal, obj, lights, spheres, triangles);
		pix[0] = 50 * light;
		pix[1] = 200 * light;
		pix[2] = 100 * light;
	}
	else{
		pix[0] = 40;
		pix[1] = 120;
		pix[2] = 240;

	}

}

void InitDrawing(char * ptr)
{
	imgptr = ptr;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&devImgPtr, XRES * YRES * 3 * sizeof(char));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devSpheres, SPHC * sizeof(Sphere));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devLights, LIGHTS * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devTriangles, TRIS * sizeof(Triangle));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}
}

void DrawFrame()
{
	InitFrame();
	
	hipError_t cudaStatus = hipMemcpy(devSpheres, spheres, SPHC * sizeof(Sphere), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	cudaStatus = hipMemcpy(devLights, lights, LIGHTS * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	cudaStatus = hipMemcpy(devTriangles, triangles, TRIS * sizeof(Triangle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	dim3 thrds(THRCOUNT, THRCOUNT);
	dim3 blocks(XRES / THRCOUNT, YRES / THRCOUNT);

	drawPixelCUDA << <blocks, thrds >> > (devImgPtr, devLights, devSpheres, devTriangles);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(imgptr, devImgPtr, XRES * YRES * 3 * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return;
	}
}
#endif
