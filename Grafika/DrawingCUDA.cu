#include "hip/hip_runtime.h"
#include "Drawing.h"
#include "Ray.h"

#ifdef CUDA

#include <math.h>
#include <Windows.h>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#define SPHC 2
#define TRIS 3
#define LIGHTS 1
#define RANDGENS 1000

#define THRCOUNT 8

//Point camera = Point(0, 0, -2.0f);
Sphere spheres[SPHC];
Light lights[LIGHTS];
Triangle triangles[TRIS];
float angle = 0;

char *imgptr, *devImgPtr;
float *realImg;

Sphere *devSpheres;
Light *devLights;
Triangle *devTriangles;
int iteration = 1;
bool started = false;
hiprandState *devState;
int fc = 0;

void InitFrame()
{
	spheres[0] = Sphere(Point(sinf(angle) * 3, -1, 8 + cosf(angle) * 3), 1);
	//spheres[0].mirror = true;

	spheres[1] = Sphere(Point(5, -1, 5), 1);
	spheres[1].color.r = 50;
	spheres[1].color.g = 200;
	spheres[1].color.b = 100;

	lights[0] = Light(Sphere(Point(-100, 100, 10), 10), .1f);
	lights[0].color.r = 239;
	lights[0].color.g = 163;
	lights[0].color.b = 56;


	//lights[1] = Sphere(Point(-7, 0, 6), 0.5);
	triangles[0] = Triangle(Point(10, -2, 0), Point(-10, -2, 0), Point(10, -2, 20));
	triangles[1] = Triangle(Point(-10, -2, 0), Point(-10, -2, 20), Point(10, -2, 20));

	triangles[2] = Triangle(Point(-4, 2, 6), Point(-5, -2, 8), Point(-5, -5, 4));
	//triangles[2].mirror = true;
	//triangles[2].color.r = 240;

	angle += 0.001f;

	hipError_t cudaStatus = hipMemcpy(devSpheres, spheres, SPHC * sizeof(Sphere), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	cudaStatus = hipMemcpy(devLights, lights, LIGHTS * sizeof(Light), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	cudaStatus = hipMemcpy(devTriangles, triangles, TRIS * sizeof(Triangle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

}

#ifdef NONRT
__device__ bool findColPointR(Ray ray, Point *colPoint, Vector *colNormal, GraphicsObject **colObj, Sphere *spheres, Triangle *triangles, Light *lights, int iterations = 1) {

	float t1, nearest = INFINITY;
	bool mirror = false;

	for (int i = 0; i < SPHC; i++) {
		if (ray.intersects(spheres[i], &t1, nullptr)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				*colPoint = ray.getPointFromT(t1);
				*colNormal = spheres[i].Normal(*colPoint);
				*colObj = spheres + i;
				mirror = spheres[i].mirror;
			}
		}
	}

	for (int i = 0; i < LIGHTS; i++) {
		if (ray.intersects(lights[i], &t1)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				*colPoint = ray.getPointFromT(t1);
				*colNormal = lights[i].Normal(*colPoint);
				*colObj = lights + i;
			}
		}
	}

	for (int i = 0; i < TRIS; i++) {
		if (ray.intersects(triangles[i], &t1)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				*colPoint = ray.getPointFromT(t1);
				*colNormal = triangles[i].n;
				*colObj = triangles + i;
				mirror = triangles[i].mirror;
			}
		}
	}

	if (mirror) {
		return findColPointR(Ray(*colPoint, ray.d.Reflect(*colNormal)), colPoint, colNormal, colObj, spheres, triangles, lights, iterations - 1);
	}

	if (nearest < INFINITY) return true;
	return false;
}

__global__ void setup_kernel(hiprandState *state) {

	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx >= RANDGENS) return;
	hiprand_init(1234 + idx, idx, 0, &state[idx]);
}

__global__ void drawPixelCUDAR(char* ptr, float* realMap, Light *lights, Sphere *spheres, Triangle *triangles, int iter, hiprandState *state) {
	int xi = blockIdx.x * THRCOUNT + threadIdx.x;
	int yi = blockIdx.y * THRCOUNT + threadIdx.y;

	if (xi > XRES || yi > YRES) return;

	float x = (xi * 2.0f + hiprand_uniform(state + ((xi * 100 + yi + 3) % RANDGENS)) * 2.0f) / YRES - XRES / (float)YRES;
	float y = (yi * 2.0f + hiprand_uniform(state + ((xi * 100 + yi + 3) % RANDGENS)) * 2.0f) / YRES - 1.0;

	char *pix = ptr + (yi * XRES + xi) * 3;
	float *rm = realMap + (yi * XRES + xi) * 3;
	 
	Point pixelPoint(x, y, 0);

	Point camera = Point(0, 0, -2.0f);
	Vector normal;
	GraphicsObject *obj = nullptr;

	Ray ray = Ray(camera, pixelPoint);

	float light;
	float ra, c1, c2, c3;

	Point colPoint;

	float expMulti = 1000;
	float rMulR = 1.0f, rMulG = 1.0f, rMulB = 1.0f;

	int bounceCount = 5;

	for (bounceCount = 5; bounceCount > 0; bounceCount--) {
		if (!findColPointR(ray, &colPoint, &normal, &obj, spheres, triangles, lights)) {

			rm[0] += 18.2 * rMulR;
			rm[1] += 42.4 * rMulG;
			rm[2] += 55.2 * rMulB;
			break;
		}
		else {
			if (obj->shape == LIGHT) {

				rm[0] += ((Light*)obj)->R() * rMulR;
				rm[1] += ((Light*)obj)->G() * rMulG;
				rm[2] += ((Light*)obj)->B() * rMulB;
				break;
			}
			else {
				rMulR *= (obj->color.r * obj->color.r) / 65100.0f;
				rMulG *= (obj->color.g * obj->color.g) / 65100.0f;
				rMulB *= (obj->color.b * obj->color.b) / 65100.0f;
				ray.o = colPoint;
				do {
					ray.d.x = hiprand_uniform(state + ((xi * 100 + yi) % RANDGENS)) * 2 - 1.0f;
					ray.d.y = hiprand_uniform(state + ((xi * 100 + yi + 1) % RANDGENS)) * 2 - 1.0f;
					ray.d.z = hiprand_uniform(state + ((xi * 100 + yi + 2) % RANDGENS)) * 2 - 1.0f;
					ray.d.Normalize();
					if (ray.d * normal <= 0) ray.d = -ray.d;
				} while (ray.d * normal <= hiprand_uniform(state + ((xi * 100 + yi + 3) % RANDGENS)));
			}
		}
	}

	c1 = sqrtf(rm[0] / iter * expMulti);
	c2 = sqrtf(rm[1] / iter * expMulti);
	c3 = sqrtf(rm[2] / iter * expMulti);

	if (c1 > 255) c1 = 255;
	if (c2 > 255) c2 = 255;
	if (c3 > 255) c3 = 255;

	pix[0] = c1;
	pix[1] = c2;
	pix[2] = c3;
	return;
}

void DrawFrame() {
	dim3 thrds(THRCOUNT, THRCOUNT);
	dim3 blocks(XRES / THRCOUNT, YRES / THRCOUNT);

	hipError_t cudaStatus;

	for (int i = 0; i < 5; i++) {
		drawPixelCUDAR << <blocks, thrds >> > (devImgPtr, realImg, devLights, devSpheres, devTriangles, iteration, devState);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			return;
		}

		iteration++;

		printf("Iteration : %d\n", iteration);
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(imgptr, devImgPtr, XRES * YRES * 3 * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return;
	}

	/*if (iteration % 50 < 5) {
		FILE* pFile;
		char name[] = "fileXX.raw";
		name[4] = fc / 10 + '0';
		name[5] = fc % 10 + '0';
		pFile = fopen(name, "wb");
		fwrite(imgptr, sizeof(char), XRES * YRES * 3, pFile);
		fclose(pFile);
		printf("Saving...\n");
		fc++;
	}*/


	/*if (iteration >= 2000) {
		iteration = 0;
		hipMemset(realImg, 0, XRES * YRES * 3 * sizeof(float));
		FILE* pFile;
		char name[] = "fileXX.raw";
		name[4] = fc / 10 + '0';
		name[5] = fc % 10 + '0';
		pFile = fopen(name, "wb");
		fwrite(imgptr, sizeof(char), XRES * YRES * 3, pFile);
		fclose(pFile);
		printf("Saving...\n");
		InitFrame();
		fc++;
	}*/
};

void InitDrawing(char *ptr) {
	imgptr = ptr;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&devImgPtr, XRES * YRES * 3 * sizeof(char));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&realImg, XRES * YRES * 3 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devSpheres, SPHC * sizeof(Sphere));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devLights, LIGHTS * sizeof(Light));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devState, sizeof(hiprandState) * RANDGENS);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devTriangles, TRIS * sizeof(Triangle));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	setup_kernel << <10, RANDGENS / 10 >> > (devState);

	InitFrame();
}

#else

__device__ float pointLit(Point &p, Vector n, GraphicsObject* self, Light *lights, Sphere *spheres, Triangle *triangles) {
	Ray ray;
	float lit = 0, t;
	bool col;
	for (int i = 0; i < LIGHTS; i++) {
		ray = Ray(p, lights[i].c);
		if (n * ray.d > 0) {
			col = false;
			for (int j = 0; j < SPHC; j++) {
				if (spheres + j != self && ray.intersects(spheres[j], &t) && t > 0.0001) {
					col = true;
					break;
				}
			}
			if (!col) {
				for (int j = 0; j < TRIS; j++) {
					if (triangles + j != self && ray.intersects(triangles[j], &t) && t > 0.0001) {
						col = true;
						break;
					}
				}
			}
			if (!col) {
				lit += n * ray.d;
			}
		}
	}
	return lit;
}

__device__ bool findColPoint(Ray ray, Point *colPoint, Vector *colNormal, GraphicsObject **colObj, Sphere *spheres, Triangle *triangles, int iterations = 2) {

	float t1, nearest = INFINITY;
	bool mirror = false;

	for (int i = 0; i < SPHC; i++) {
		if (ray.intersects(spheres[i], &t1, nullptr)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				*colPoint = ray.getPointFromT(t1);
				*colNormal = spheres[i].Normal(*colPoint);
				*colObj = spheres + i;
				mirror = spheres[i].mirror;
			}
		}
	}

	for (int i = 0; i < TRIS; i++) {
		if (ray.intersects(triangles[i], &t1)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				*colPoint = ray.getPointFromT(t1);
				*colNormal = triangles[i].n;
				*colObj = triangles + i;
				mirror = triangles[i].mirror;
			}
		}
	}

	if (mirror && iterations > 0) {
		return findColPoint(Ray(*colPoint, ray.d.Reflect(*colNormal)), colPoint, colNormal, colObj, spheres, triangles, iterations - 1);
	}

	if (nearest < INFINITY) return true;
	return false;
}


__global__ void drawPixelCUDA(char* ptr, Light *lights, Sphere *spheres, Triangle *triangles) {
	int xi = blockIdx.x * THRCOUNT + threadIdx.x;
	int yi = blockIdx.y * THRCOUNT + threadIdx.y;

	if (xi > XRES || yi > YRES) return;

	float x = xi * 2.0f / YRES - XRES / (float)YRES;
	float y = yi * 2.0 / YRES - 1.0;

	char *pix = ptr + (yi * XRES + xi) * 3;

	Point pixelPoint(x, y, 0);

	Point camera = Point(0, 0, -2.0f);
	Vector normal;
	GraphicsObject *obj;

	Ray ray = Ray(camera, pixelPoint);

	float light;

	Point colPoint;

	if (findColPoint(ray, &colPoint, &normal, &obj, spheres, triangles)) {
		light = pointLit(colPoint, normal, obj, lights, spheres, triangles);
		pix[0] = obj->color.r * light + 8 * (1 - light);
		pix[1] = obj->color.g * light + 24 * (1 - light);
		pix[2] = obj->color.b * light + 48 * (1 - light);
	}
	else{
		pix[0] = 40;
		pix[1] = 120;
		pix[2] = 240;

	}

}

void InitDrawing(char * ptr)
{
	imgptr = ptr;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&devImgPtr, XRES * YRES * 3 * sizeof(char));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devSpheres, SPHC * sizeof(Sphere));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devLights, LIGHTS * sizeof(Light));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devTriangles, TRIS * sizeof(Triangle));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}
	started = true;
}

void DrawFrame()
{
	if (!started) return;
	InitFrame();
	
	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(devSpheres, spheres, SPHC * sizeof(Sphere), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	cudaStatus = hipMemcpy(devLights, lights, LIGHTS * sizeof(Light), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	cudaStatus = hipMemcpy(devTriangles, triangles, TRIS * sizeof(Triangle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	dim3 thrds(THRCOUNT, THRCOUNT);
	dim3 blocks(XRES / THRCOUNT, YRES / THRCOUNT);

	drawPixelCUDA << <blocks, thrds >> > (devImgPtr, devLights, devSpheres, devTriangles);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(imgptr, devImgPtr, XRES * YRES * 3 * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return;
	}
}

#endif

#endif
