#include "hip/hip_runtime.h"
#include "Drawing.h"

#ifdef CUDA

#include <math.h>
#include <Windows.h>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

/*#define SPHC 2
#define TRIS 3
#define LIGHTS 1*/
#define RANDGENS 1000

#define THRCOUNT 8

float angle = 0;

char *imgptr, *devImgPtr;
float *realImg = nullptr;

int iteration = 1;
bool started = false;
hiprandState *devState;
int fc = 0;

SceneData sd, devSdCopy;
SceneData *devSd;

Light *devLights;
Sphere *devSpheres;
Triangle *devTriangles;
Material *devMaterials;

void InitFrame()
{

	sd.genCameraCoords();
	devSdCopy = sd.genDeviceData(devSpheres, devTriangles, devLights, devMaterials);

	hipError_t cudaStatus = hipMemcpy(devSpheres, sd.spheres, sd.nSpheres * sizeof(Sphere), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	cudaStatus = hipMemcpy(devLights, sd.lights, sd.nLights * sizeof(Light), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	cudaStatus = hipMemcpy(devTriangles, sd.triangles, sd.nTriangles * sizeof(Triangle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	cudaStatus = hipMemcpy(devMaterials, sd.materials, sd.nMaterials * sizeof(Material), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	cudaStatus = hipMemcpy(devSd, &devSdCopy, sizeof(SceneData), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return;
	}

	if (sd.reset) {
		sd.reset = false;
		iteration = 1;

		if(realImg != nullptr)
			hipMemset(realImg, 0, XRES * YRES * 3 * sizeof(float));
	}
}

__device__ ColorReal traceRand(Ray ray, SceneData *sd, hiprandState *state, int iterations = 20) {
	float t1, nearest = INFINITY;
	ColorReal colorMultiplier(1, 1, 1);
	ColorReal colGet;
	Point colPoint;
	Vector colNormal;
	GraphicsObject *colObj;

	if (iterations <= 0) {
		return ColorReal(0, 0, 0);
	}

	for (int i = 0; i < sd->nSpheres; i++) {
		if (ray.intersects(sd->spheres[i], &colGet, &t1, nullptr)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				colPoint = ray.getPointFromT(t1);
				colNormal = sd->spheres[i].Normal(colPoint);
				colObj = sd->spheres + i;
				colorMultiplier = colGet.getColorIntesity(sd->gamma);
			}
		}
	}

	for (int i = 0; i < sd->nLights; i++) {
		if (ray.intersects(sd->lights[i], &colGet, &t1)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				colPoint = ray.getPointFromT(t1);
				colNormal = sd->lights[i].Normal(colPoint);
				colObj = sd->lights + i;
				colorMultiplier = colGet.getColorIntesity(sd->gamma) * sd->lights[i].intenisty;
			}
		}
	}

	for (int i = 0; i < sd->nTriangles; i++) {
		if (ray.intersects(sd->triangles[i], &colGet, &t1)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				colPoint = ray.getPointFromT(t1);
				colNormal = sd->triangles[i].n;
				colObj = sd->triangles + i;
				colorMultiplier = colGet.getColorIntesity(sd->gamma);
			}
		}
	}

	if (nearest == INFINITY) {
		return sd->ambient.mat.color.getColorIntesity(sd->gamma) * sd->ambient.intenisty;
	}
	else if (colObj->shape == LIGHT) {
		return colorMultiplier;
	}
	else {
		if (colObj->mat.mirror) {
			return colorMultiplier *= traceRand(Ray(colPoint, ray.d.Reflect(colNormal)), sd, state, iterations - 1);
		}
		else if (colObj->mat.transparent) {
			return colorMultiplier *= traceRand(Ray(colPoint, ray.d.Refract(colNormal, colObj->mat.refIndex)), sd, state, iterations - 1);
		}
		else {
			ray.o = colPoint;
			if (ray.d * colNormal > 0) colNormal = -colNormal;
			do {
				ray.d.x = hiprand_uniform(state) * 2 - 1.0f;
				ray.d.y = hiprand_uniform(state) * 2 - 1.0f;
				ray.d.z = hiprand_uniform(state) * 2 - 1.0f;
				ray.d.Normalize();
				if (ray.d * colNormal <= 0) ray.d = -ray.d;
			} while (ray.d * colNormal <= hiprand_uniform(state));
			return colorMultiplier *= traceRand(ray, sd, state, iterations - 1);
		}
	}
}

__device__ bool findColPoint(Ray ray, Point *colPoint, Vector *colNormal, GraphicsObject **colObj, SceneData *sd, int iterations = 3) {

	float t1, nearest = INFINITY;
	bool mirror = false;
	bool transparent = false;

	for (int i = 0; i < sd->nSpheres; i++) {
		if (ray.intersects(sd->spheres[i], nullptr, &t1, nullptr)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				*colPoint = ray.getPointFromT(t1);
				*colNormal = sd->spheres[i].Normal(*colPoint);
				*colObj = sd->spheres + i;
				mirror = (*colObj)->mat.mirror;
				transparent = (*colObj)->mat.transparent;
			}
		}
	}

	for (int i = 0; i < sd->nLights; i++) {
		if (ray.intersects(sd->lights[i], nullptr, &t1)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				*colPoint = ray.getPointFromT(t1);
				*colNormal = sd->lights[i].Normal(*colPoint);
				*colObj = sd->lights + i;
			}
		}
	}

	for (int i = 0; i < sd->nTriangles; i++) {
		if (ray.intersects(sd->triangles[i], nullptr, &t1)) {
			if (t1 < nearest && t1 > 0.001) {
				nearest = t1;
				*colPoint = ray.getPointFromT(t1);
				*colNormal = sd->triangles[i].n;
				*colObj = sd->triangles + i;
				mirror = (*colObj)->mat.mirror;
				transparent = (*colObj)->mat.transparent;
			}
		}
	}

	if (mirror && iterations > 0) {
		return findColPoint(Ray(*colPoint, ray.d.Reflect(*colNormal)), colPoint, colNormal, colObj, sd, iterations - 1);
	}
	else if(transparent && iterations > 0){
		return findColPoint(Ray(*colPoint, ray.d.Refract(*colNormal, (*colObj)->mat.refIndex)), colPoint, colNormal, colObj, sd, iterations - 1);
	}

	if (nearest < INFINITY) return true;
	return false;
}

__global__ void setup_kernel(hiprandState *state) {

	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx >= RANDGENS) return;
	hiprand_init(1234 + idx, idx, 0, &state[idx]);
}

__global__ void drawPixelCUDAR(char* ptr, float* realMap, SceneData *sd, int iter, hiprandState *state) {
	int xi = blockIdx.x * THRCOUNT + threadIdx.x;
	int yi = blockIdx.y * THRCOUNT + threadIdx.y;

	if (xi > XRES || yi > YRES) return;

	float x = (xi * 2.0f + hiprand_uniform(state + ((xi * 100 + yi + 3) % RANDGENS)) * 2.0f) / YRES - XRES / (float)YRES;
	float y = (yi * 2.0f + hiprand_uniform(state + ((xi * 100 + yi + 3) % RANDGENS)) * 2.0f) / YRES - 1.0;

	Color *pix = (Color*)(ptr + (yi * XRES + xi) * 3);
	ColorReal *rm = (ColorReal*)(realMap + (yi * XRES + xi) * 3);

	//Point pixelPoint = Point(10 + x, y, 0);

	Point pixelPoint = sd->camera + sd->c2S + sd->sR * x + sd->sD * y;

	float focalDistance = sd->focalDistance;

	Vector normal;
	GraphicsObject *obj = nullptr;

	Ray ray = Ray(sd->camera, pixelPoint);

	if (sd->dofStr > 0.000001f) {
		Point focalPoint = sd->camera + (Vector)(pixelPoint - sd->camera) * (1 + focalDistance / sd->camDist);

		float pointMove = sd->dofStr, xOff, yOff;

		float ang = hiprand_uniform(state + ((xi * 100 + yi) % RANDGENS)) * 6.28315f;
		pointMove *= hiprand_uniform(state + ((xi * 100 + yi) % RANDGENS));
		xOff = sinf(ang) * sqrtf(pointMove);
		yOff = cosf(ang) * sqrtf(pointMove);
		/*do {
			xOff = (hiprand_uniform(state + ((xi * 100 + yi) % RANDGENS)) * 2 - 1.0f) * pointMove;
			yOff = (hiprand_uniform(state + ((xi * 100 + yi) % RANDGENS)) * 2 - 1.0f) * pointMove;
		} while (sqrtf(xOff * xOff + yOff * yOff) > pointMove);*/
		Point passPoint = pixelPoint + sd->sR * xOff + sd->sD * yOff;
		ray = Ray(passPoint, focalPoint);
	}

	float light;
	float ra, c1, c2, c3;

	Point colPoint;

	*rm += traceRand(ray, sd, state + ((xi * XRES + yi + 3) + (iter* 123)) % RANDGENS, sd->bounces);
	*pix = rm->getPixColor(sd->gamma, sd->expMultiplier / iter);

	return;
}

__device__ float pointLit(Point &p, Vector n, GraphicsObject* self, SceneData *sd) {
	Ray ray;
	float lit = 0, t;
	bool col;
	for (int i = 0; i < sd->nLights; i++) {
		ray = Ray(p, sd->lights[i].c);
		if (n * ray.d > 0) {
			col = false;
			for (int j = 0; j < sd->nSpheres; j++) {
				if (sd->spheres + j != self && ray.intersects(sd->spheres[j], nullptr, &t) && t > 0.0001) {
					col = true;
					break;
				}
			}
			if (!col) {
				for (int j = 0; j < sd->nTriangles; j++) {
					if (sd->triangles + j != self && ray.intersects(sd->triangles[j], nullptr, &t) && t > 0.0001) {
						col = true;
						break;
					}
				}
			}
			if (!col) {
				lit += n * ray.d;
			}
		}
	}
	return lit;
}

__global__ void drawPixelCUDA(char* ptr, SceneData *sd) {
	int xi = blockIdx.x * THRCOUNT + threadIdx.x;
	int yi = blockIdx.y * THRCOUNT + threadIdx.y;

	if (xi > XRES || yi > YRES) return;

	float x = xi * 2.0f / YRES - XRES / (float)YRES;
	float y = yi * 2.0 / YRES - 1.0;

	Color *pix = (Color*)(ptr + (yi * XRES + xi) * 3);

	Point pixelPoint = sd->camera + sd->c2S + sd->sR * x + sd->sD * y;

	ColorReal color;

	Vector normal;
	GraphicsObject *obj;

	Ray ray = Ray(sd->camera, pixelPoint);

	float light = 1.0f;

	Point colPoint;

	if (findColPoint(ray, &colPoint, &normal, &obj, sd)) {
		if (obj->shape == LIGHT) light == 1.0f;
		else light = pointLit(colPoint, normal, obj, sd);

		if (obj->shape == TRIANGLE && ((Triangle*)obj)->mat.texture.width != 0) {
			float coords[] = { 0, 0 };
			((Triangle*)obj)->interpolatePoint(colPoint, (float*)&(((Triangle*)obj)->t0), (float*)&(((Triangle*)obj)->t1), (float*)&(((Triangle*)obj)->t2), coords, 2);
			ColorReal c = obj->mat.getColor(coords[0], coords[1]);

			color = c * light;
		}
		else {
			color = obj->mat.getColor(0, 0) * light;
		}
	}
	else{
		color = sd->ambient.mat.color;
	}
	*pix = color.getPixColor();
}

void InitDrawing(char * ptr)
{
	imgptr = ptr;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&devImgPtr, XRES * YRES * 3 * sizeof(char));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&realImg, XRES * YRES * 3 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devSpheres, sd.nSpheres * sizeof(Sphere));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devLights, sd.nLights * sizeof(Light));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devTriangles, sd.nTriangles * sizeof(Triangle));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devMaterials, sd.nMaterials * sizeof(Material));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devSd, sizeof(SceneData));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	cudaStatus = hipMalloc((void**)&devState, sizeof(hiprandState) * RANDGENS);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return;
	}

	setup_kernel << <10, RANDGENS / 10 >> > (devState);

	InitFrame();

}

void DrawFrame()
{
	if (sd.realTime) {
		InitFrame();

		hipError_t cudaStatus;

		dim3 thrds(THRCOUNT, THRCOUNT);
		dim3 blocks(XRES / THRCOUNT, YRES / THRCOUNT);

		drawPixelCUDA << <blocks, thrds >> > (devImgPtr, devSd);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			return;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(imgptr, devImgPtr, XRES * YRES * 3 * sizeof(char), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return;
		}
	}
	else {
		dim3 thrds(THRCOUNT, THRCOUNT);
		dim3 blocks(XRES / THRCOUNT, YRES / THRCOUNT);

		hipError_t cudaStatus;

		drawPixelCUDAR << <blocks, thrds >> > (devImgPtr, realImg, devSd, iteration, devState);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			return;
		}

		iteration++;

		printf("Iteration : %d\n", iteration);

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(imgptr, devImgPtr, XRES * YRES * 3 * sizeof(char), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return;
		}

		/*if (iteration >= 2000) {
			iteration = 0;
			hipMemset(realImg, 0, XRES * YRES * 3 * sizeof(float));
			FILE* pFile;
			char name[] = "fileXX.raw";
			name[4] = fc / 10 + '0';
			name[5] = fc % 10 + '0';
			pFile = fopen(name, "wb");
			fwrite(imgptr, sizeof(char), XRES * YRES * 3, pFile);
			fclose(pFile);
			printf("Saving...\n");
			InitFrame();
			fc++;
		}*/
	}
}

DEVICE_PREFIX void SceneData::genCameraCoords()
{
	if (camXang > 6.28318f) camXang -= 6.28318f;
	if (camXang < 0.0f) camXang += 6.28318f;
	if (camYang >= 1.5707f && camYang < 3.14159f) camYang = 1.57f;
	else {
		if (camYang < 0.0f) camYang += 6.28318f;
		//TODO doesnt work
		if (camYang > 3.141592f && camYang <= 4.712388f) camYang = 4.714f;
	}

	c2S = Vector(0, 0, 1);

	c2S = Vector(-sinf(camXang), tanf(camYang), cosf(camXang));

	c2S = c2S.Normalize() * camDist;

	sR = Vector(cosf(camXang), 0, sinf(camXang));

	sD = (c2S / camDist) % sR;

}

DEVICE_PREFIX SceneData SceneData::genDeviceData(Sphere *devS, Triangle *devTr, Light *devL, Material *devMa)
{
	SceneData ret = *this;
	/*for (int i = 0; i < nTriangles; i++) {
		ret.triangles[i]. = devTe + triangles[i].texIndex;
	}*/

	ret.lights = devL;
	ret.triangles = devTr;
	ret.spheres = devS;
	ret.materials = devMa;
	return ret;
}

void SceneData::assignPointersHost() {};

#include "Point.cpp"
#include "Ray.cpp"
#include "Sphere.cpp"
#include "Texture.cpp"
#include "Triangle.cpp"
#include "Vector.cpp"
#include "Color.cpp"
#include "Material.cpp"

#endif
